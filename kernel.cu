#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <ctime>
#include <chrono>
#include <iomanip>
#include <fstream>

#include "personnel.hpp"
#include "kernel.h"

__forceinline__ __device__ float diff_of_products(int a, int b, int c, int d) {
    int w = d * c;
    int e = fmaf(-d, c, w);
    int f = fmaf(a, b, -w);
    return f + e;
}

__device__ int area2CUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool xorBoolCUDA(bool x, bool y) {
    return !x ^ !y;
}

__device__ bool intersectPropCUDA(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy) {
    if (collinearCUDA(ax, ay, bx, by, cx, cy) || collinearCUDA(ax, ay, bx, by, dx, dy) || collinearCUDA(cx, cy, dx, dy, ax, ay) || collinearCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftCUDA(ax, ay, bx, by, cx, cy), leftCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftCUDA(cx, cy, dx, dy, ax, ay), leftCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentCUDA(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy, bool& isIllegal) {
    if (intersectPropCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenCUDA(ax, ay, bx, by, cx, cy) || betweenCUDA(ax, ay, bx, by, dx, dy) || betweenCUDA(cx, cy, dx, dy, ax, ay) || betweenCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__forceinline__ __device__ float diff_of_productsReel(double a, double b, double c, double d) {
    float w = d * c;
    float e = fmaf(-d, c, w);
    float f = fmaf(a, b, -w);
    return f + e;
}

__device__ double area2ReelCUDA(double ax, double ay, double bx, double by, double cx, double cy) {
    //return diff_of_productsReel((bx - ax), (cy - ay), (cx - ax), (by - ay));
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftReelCUDA(double ax, double ay, double bx, double by, double cx, double cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearReelCUDA(double ax, double ay, double bx, double by, double cx, double cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool intersectPropReelCUDA(double ax, double ay, double bx, double by, double cx, double cy, double dx, double dy) {
    if (collinearReelCUDA(ax, ay, bx, by, cx, cy) || collinearReelCUDA(ax, ay, bx, by, dx, dy) || collinearReelCUDA(cx, cy, dx, dy, ax, ay) || collinearReelCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftReelCUDA(ax, ay, bx, by, cx, cy), leftReelCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftReelCUDA(cx, cy, dx, dy, ax, ay), leftReelCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenReelCUDA(double ax, double ay, double bx, double by, double cx, double cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentReelCUDA(double ax, double ay, double bx, double by, double cx, double cy, double dx, double dy, bool& isIllegal) {
    if (intersectPropReelCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenReelCUDA(ax, ay, bx, by, cx, cy) || betweenReelCUDA(ax, ay, bx, by, dx, dy) || betweenReelCUDA(cx, cy, dx, dy, ax, ay) || betweenReelCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentReelCUDA(double ax, double ay, double bx, double by, double cx, double cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__global__ void kernelUpdateCrossings(int* nodes, int* edges, long* scores, int* newCoords, int* nodeId, int numThreads,int numNodes, int numEdges, int* commonNodeEdges) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > numThreads) { return; }
    int id = nodeId[i]*2;
    int newX = newCoords[i*2];
    int newY = newCoords[i*2+1];
    long score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    int commonNodex, commonNodey;
    for (int i = 0; i < numEdges-1; i+=2) {
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1*2];
        node1y = nodes[nodeIndex1*2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2*2];
        node2y = nodes[nodeIndex2*2 + 1];
        for (int j = i + 1; j < numEdges; j+=2) {
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3*2];
            node3y = nodes[nodeIndex3*2 + 1];
            nodeIndex4 = edges[j+1];
            node4x = nodes[nodeIndex4*2];
            node4y = nodes[nodeIndex4*2 + 1];
            bool isIllegal = false;
            if (commonNodeEdges[i*numNodes+j] == -1) {
                if (seCroisentCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000; }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[j*numNodes+i];
                commonNodex = nodes[commonNodeIndex * 2];
                commonNodey = nodes[commonNodeIndex * 2 + 1];
                if (surSegmentCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[i*numNodes+j];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegmentCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateCrossingsReel(double* nodes, int* edges, long* scores, double* newCoords, int* nodeId, int numThreads, int numNodes, int numEdges, int* commonNodeEdges) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int id = nodeId[i] * 2;
    double newX, newY;
    if (i == 0) {
        newX = nodes[id];
        newY = nodes[id + 1];
    }
    else {
        newX = newCoords[i * 2];
        newY = newCoords[i * 2 + 1];
    }
    long score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int areteIndex1, areteIndex2;
    double node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    double commonNodex, commonNodey;
    int limiteIndex1 = (numEdges * 2) - 2;
    int limiteIndex2 = (numEdges * 2);
    for (int i = 0; i < limiteIndex1; i += 2) {
        areteIndex1 = i / 2;
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1 * 2];
        node1y = nodes[nodeIndex1 * 2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2 * 2];
        node2y = nodes[nodeIndex2 * 2 + 1];
        if (nodeIndex1 == id) {
            node1x = newX;
            node1y = newY;
        }
        else if (nodeIndex2 == id) {
            node2x = newX;
            node2y = newY;
        }
        for (int j = i + 2; j < limiteIndex2; j += 2) {
            areteIndex2 = j / 2;
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3 * 2];
            node3y = nodes[nodeIndex3 * 2 + 1];
            nodeIndex4 = edges[j + 1];
            node4x = nodes[nodeIndex4 * 2];
            node4y = nodes[nodeIndex4 * 2 + 1];
            bool isIllegal = false;
            if (nodeIndex3 == id) {
                node3x = newX;
                node3y = newY;
            }
            else if (nodeIndex4 == id) {
                node4x = newX;
                node4y = newY;
            }
            if (commonNodeEdges[areteIndex1 * numEdges + areteIndex2] == -1) {
                if (seCroisentReelCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000;  }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[areteIndex2 * numEdges + areteIndex1];
                if (commonNodeIndex == id) {
                    commonNodex = newX;
                    commonNodey = newY;
                }
                else {
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                }
                if (surSegmentReelCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[areteIndex1 * numEdges + areteIndex2];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegmentReelCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

extern "C" void rechercheTabouGPU(const int* nodes, const int* edges, long* scores, const int* newCoords, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int blockSize, const int gridSize) {
    int* devNodes;
    int* devEdges;
    long* devScores;
    int* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(int) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(long) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(int) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(int) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(long) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(int) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);

    // Update crossing score
    kernelUpdateCrossings<<<gridSize, blockSize>>>(devNodes,devEdges,devScores,devNewCoords,devNodeId,numThreads,numNodes,numEdges,devCommonNodeEdges);
    // Update vector device to host
    hipMemcpy(scores, devScores, sizeof(long) * numThreads, hipMemcpyDeviceToHost);

    for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }

    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
}

extern "C" void rechercheTabouGPUReel(const double* nodes, const int* edges, long* scores, const double* newCoords, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int blockSize, const int gridSize) {
    double* devNodes;
    int* devEdges;
    long* devScores;
    double* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(double) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(long) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(double) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);
    hipMalloc((void**)&devCommonNodeEdges, sizeof(int) * numEdges * numEdges);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(double) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(long) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(double) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devCommonNodeEdges, commonNodeEdges, sizeof(int) * numEdges * numEdges, hipMemcpyHostToDevice);

    for (int iter = 0; iter < 1; iter++) {
        // Update crossing score
        auto start = std::chrono::system_clock::now();
        kernelUpdateCrossingsReel<<<gridSize, blockSize>>>(devNodes, devEdges, devScores, devNewCoords, devNodeId, numThreads, numNodes, numEdges, devCommonNodeEdges);
        // Update vector device to host
        hipMemcpy(scores, devScores, sizeof(long) * numThreads, hipMemcpyDeviceToHost);
        auto end = std::chrono::system_clock::now();
        std::chrono::duration<double> secondsTotal = end - start;
        std::string nomFichier = chemin + "/resultats/cuda.csv";
        std::ofstream resultats(nomFichier, std::ios_base::app);
        resultats << std::fixed << std::setprecision(8) << gridSize << "," << blockSize << "," << secondsTotal.count() << std::endl;
        resultats.close();
    }

    for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }
    //std::cout << "score: " << scores[0] << std::endl;
    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
    hipFree(devCommonNodeEdges);
}