#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""

#include "kernel.h"

__device__ int area2(int ax, int ay, int bx, int by, int cx, int cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool left(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinear(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool xorBool(bool x, bool y) {
    return !x ^ !y;
}

__device__ bool intersectProp(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy) {
    if (collinear(ax, ay, bx, by, cx, cy) || collinear(ax, ay, bx, by, dx, dy) || collinear(cx, cy, dx, dy, ax, ay) || collinear(cx, cy, dx, dy, bx, by))
        return false;
    return xorBool(left(ax, ay, bx, by, cx, cy), left(ax, ay, bx, by, dx, dy)) && xorBool(left(cx, cy, dx, dy, ax, ay), left(cx, cy, dx, dy, bx, by));
}

__device__ bool between(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinear(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisent(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy, bool& isIllegal) {
    if (intersectProp(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (between(ax, ay, bx, by, cx, cy) || between(ax, ay, bx, by, dx, dy) || between(cx, cy, dx, dy, ax, ay) || between(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegment(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinear(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ float area2Reel(float ax, float ay, float bx, float by, float cx, float cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftReel(float ax, float ay, float bx, float by, float cx, float cy) {
    return area2Reel(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearReel(float ax, float ay, float bx, float by, float cx, float cy) {
    return area2Reel(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool intersectPropReel(float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy) {
    if (collinearReel(ax, ay, bx, by, cx, cy) || collinearReel(ax, ay, bx, by, dx, dy) || collinearReel(cx, cy, dx, dy, ax, ay) || collinearReel(cx, cy, dx, dy, bx, by))
        return false;
    return xorBool(leftReel(ax, ay, bx, by, cx, cy), leftReel(ax, ay, bx, by, dx, dy)) && xorBool(leftReel(cx, cy, dx, dy, ax, ay), leftReel(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenReel(float ax, float ay, float bx, float by, float cx, float cy) {
    if (!collinearReel(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentReel(float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy, bool& isIllegal) {
    if (intersectPropReel(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenReel(ax, ay, bx, by, cx, cy) || betweenReel(ax, ay, bx, by, dx, dy) || betweenReel(cx, cy, dx, dy, ax, ay) || betweenReel(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentReel(float ax, float ay, float bx, float by, float cx, float cy) {
    if (!collinearReel(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__global__ void kernelUpdateCrossings(int* nodes, int* edges, long* scores, int* newCoords, int* nodeId, int numThreads,int numNodes, int numEdges, int* commonNodeEdges) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > numThreads) { return; }
    int id = nodeId[i]*2;
    int newX = newCoords[i*2];
    int newY = newCoords[i*2+1];
    long score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    int commonNodex, commonNodey;
    for (int i = 0; i < numEdges-1; i+=2) {
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1*2];
        node1y = nodes[nodeIndex1*2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2*2];
        node2y = nodes[nodeIndex2*2 + 1];
        for (int j = i + 1; j < numEdges; j+=2) {
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3*2];
            node3y = nodes[nodeIndex3*2 + 1];
            nodeIndex4 = edges[j+1];
            node4x = nodes[nodeIndex4*2];
            node4y = nodes[nodeIndex4*2 + 1];
            bool isIllegal = false;
            if (commonNodeEdges[i*numNodes+j] == -1) {
                if (seCroisent(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000; }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[j*numNodes+i];
                commonNodex = nodes[commonNodeIndex * 2];
                commonNodey = nodes[commonNodeIndex * 2 + 1];
                if (surSegment(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[i*numNodes+j];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegment(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateCrossingsReel(float* nodes, int* edges, long* scores, float* newCoords, int* nodeId, int numThreads, int numNodes, int numEdges, int* commonNodeEdges) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > numThreads) { return; }
    int id = nodeId[i] * 2;
    float newX = newCoords[i * 2];
    float newY = newCoords[i * 2 + 1];
    long score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    float node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    float commonNodex, commonNodey;
    for (int i = 0; i < numEdges - 1; i += 2) {
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1 * 2];
        node1y = nodes[nodeIndex1 * 2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2 * 2];
        node2y = nodes[nodeIndex2 * 2 + 1];
        for (int j = i + 1; j < numEdges; j += 2) {
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3 * 2];
            node3y = nodes[nodeIndex3 * 2 + 1];
            nodeIndex4 = edges[j + 1];
            node4x = nodes[nodeIndex4 * 2];
            node4y = nodes[nodeIndex4 * 2 + 1];
            bool isIllegal = false;
            if (commonNodeEdges[i*numNodes+j] == -1) {
                if (seCroisentReel(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000; }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[j*numNodes+i];
                commonNodex = nodes[commonNodeIndex * 2];
                commonNodey = nodes[commonNodeIndex * 2 + 1];
                if (surSegmentReel(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[i*numNodes+j];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegmentReel(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

extern "C" void rechercheTabouGPU(const int* nodes, const int* edges, long* scores, const int* newCoords, const int* nodeId, const int numNodes, const int numEdges, const int blockSize, const int gridSize, const int* commonNodeEdges) {
    int* devNodes;
    int* devEdges;
    long* devScores;
    int* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(int) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(long) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(int) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(int) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(long) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(int) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);

    // Update crossing score
    kernelUpdateCrossings<<<gridSize, blockSize>>>(devNodes,devEdges,devScores,devNewCoords,devNodeId,numThreads,numNodes,numEdges,devCommonNodeEdges);
    // Update vector device to host
    hipMemcpy(scores, devScores, sizeof(long) * numThreads, hipMemcpyDeviceToHost);

    for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }

    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
}

extern "C" void rechercheTabouGPUReel(const float* nodes, const int* edges, long* scores, const float* newCoords, const int* nodeId, const int numNodes, const int numEdges, const int blockSize, const int gridSize, const int* commonNodeEdges) {
    float* devNodes;
    int* devEdges;
    long* devScores;
    float* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(float) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(long) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(float) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);
    hipMalloc((void**)&devCommonNodeEdges, sizeof(int) * numNodes * numNodes);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(float) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(long) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(float) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devCommonNodeEdges, nodeId, sizeof(int) * numNodes * numNodes, hipMemcpyHostToDevice);

    // Update crossing score
    kernelUpdateCrossingsReel<<<gridSize, blockSize>>>(devNodes, devEdges, devScores, devNewCoords, devNodeId, numThreads,numNodes, numEdges, devCommonNodeEdges);
    // Update vector device to host
    hipMemcpy(scores, devScores, sizeof(long) * numThreads, hipMemcpyDeviceToHost);

    for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }

    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
    hipFree(devCommonNodeEdges);
}