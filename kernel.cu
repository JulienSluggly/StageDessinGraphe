#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <ctime>
#include <chrono>
#include <iomanip>
#include <fstream>

#include "personnel.hpp"
#include "utilitaire.hpp"
#include "kernel.h"

__device__ int area2CUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool xorBoolCUDA(const bool x, const bool y) {
    return !x ^ !y;
}

__device__ bool intersectPropCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy, const int dx, const int dy) {
    if (collinearCUDA(ax, ay, bx, by, cx, cy) || collinearCUDA(ax, ay, bx, by, dx, dy) || collinearCUDA(cx, cy, dx, dy, ax, ay) || collinearCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftCUDA(ax, ay, bx, by, cx, cy), leftCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftCUDA(cx, cy, dx, dy, ax, ay), leftCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy, const int dx, const int dy, bool& isIllegal) {
    if (intersectPropCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenCUDA(ax, ay, bx, by, cx, cy) || betweenCUDA(ax, ay, bx, by, dx, dy) || betweenCUDA(cx, cy, dx, dy, ax, ay) || betweenCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentCUDA(const int ax, const int ay, const int bx, const int by, const int cx, const int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ float area2ReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool intersectPropReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy, const float dx, const float dy) {
    if (collinearReelCUDA(ax, ay, bx, by, cx, cy) || collinearReelCUDA(ax, ay, bx, by, dx, dy) || collinearReelCUDA(cx, cy, dx, dy, ax, ay) || collinearReelCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftReelCUDA(ax, ay, bx, by, cx, cy), leftReelCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftReelCUDA(cx, cy, dx, dy, ax, ay), leftReelCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy, const float dx, const float dy, bool& isIllegal) {
    if (intersectPropReelCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenReelCUDA(ax, ay, bx, by, cx, cy) || betweenReelCUDA(ax, ay, bx, by, dx, dy) || betweenReelCUDA(cx, cy, dx, dy, ax, ay) || betweenReelCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentReelCUDA(const float ax, const float ay, const float bx, const float by, const float cx, const float cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__global__ void kernelUpdateCrossings(int* nodes, const int* edges, int* slots, const int* slotsCoord, int* scores, int* newSlotsPos, const int* nodeId, const int numThreads, const int numNodes, const int numEdges, const int numSlots, const int* commonNodeEdges, const int PENALITE_MAX, const int PENALITE_MAX_SELF) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int firstNodeId = nodeId[i];
    int secondNodeId = -1;
    int newX1, newY1, newX2, newY2;
    hiprandState state;
    hiprand_init(clock(), i, 0, &state);
    if (i != 0) {
        int randomSlot = ceil(hiprand_uniform(&state) * (numSlots-1));
        newSlotsPos[i] = randomSlot;
        if (slots[randomSlot] != -1) {
            secondNodeId = slots[randomSlot];
            int oldSlotId = nodes[firstNodeId];
            newX2 = slotsCoord[oldSlotId * 2];
            newY2 = slotsCoord[oldSlotId * 2 + 1];
        }
        newX1 = slotsCoord[randomSlot * 2];
        newY1 = slotsCoord[randomSlot * 2 + 1];
    }
    int score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int areteIndex1, areteIndex2;
    int node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    int commonNodex, commonNodey;
    int limiteIndex1 = (numEdges * 2) - 2;
    int limiteIndex2 = (numEdges * 2);
    for (int i = 0; i < limiteIndex1; i += 2) {
        areteIndex1 = i / 2;
        nodeIndex1 = edges[i];
        node1x = slotsCoord[nodes[nodeIndex1] * 2];
        node1y = slotsCoord[nodes[nodeIndex1] * 2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = slotsCoord[nodes[nodeIndex2] * 2];
        node2y = slotsCoord[nodes[nodeIndex2] * 2 + 1];
        if (nodeIndex1 == firstNodeId) {
            node1x = newX1;
            node1y = newY1;
        }
        else if (nodeIndex1 == secondNodeId) {
            node1x = newX2;
            node1y = newY2;
        }
        if (nodeIndex2 == firstNodeId) {
            node2x = newX1;
            node2y = newY1;
        }
        else if (nodeIndex2 == secondNodeId) {
            node2x = newX2;
            node2y = newY2;
        }
        for (int j = i + 2; j < limiteIndex2; j += 2) {
            areteIndex2 = j / 2;
            nodeIndex3 = edges[j];
            node3x = slotsCoord[nodes[nodeIndex3] * 2];
            node3y = slotsCoord[nodes[nodeIndex3] * 2 + 1];
            nodeIndex4 = edges[j + 1];
            node4x = slotsCoord[nodes[nodeIndex4] * 2];
            node4y = slotsCoord[nodes[nodeIndex4] * 2 + 1];
            bool isIllegal = false;
            if (nodeIndex3 == firstNodeId) {
                node3x = newX1;
                node3y = newY1;
            }
            else if (nodeIndex3 == secondNodeId) {
                node3x = newX2;
                node3y = newY2;
            }
            if (nodeIndex4 == firstNodeId) {
                node4x = newX1;
                node4y = newY1;
            }
            else if (nodeIndex4 == secondNodeId) {
                node4x = newX2;
                node4y = newY2;
            }
            if (commonNodeEdges[areteIndex1 * numEdges + areteIndex2] == -1) {
                if (seCroisentCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += PENALITE_MAX; }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[areteIndex2 * numEdges + areteIndex1];
                if (commonNodeIndex == firstNodeId) {
                    commonNodex = newX1;
                    commonNodey = newY1;
                }
                else if (commonNodeIndex == secondNodeId) {
                    commonNodex = newX2;
                    commonNodey = newY2;
                }
                else {
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                }
                if (surSegmentCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += PENALITE_MAX_SELF; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[areteIndex1 * numEdges + areteIndex2];
                    if (commonNodeIndex == firstNodeId) {
                        commonNodex = newX1;
                        commonNodey = newY1;
                    }
                    else if (commonNodeIndex == secondNodeId) {
                        commonNodex = newX2;
                        commonNodey = newY2;
                    }
                    else {
                        commonNodex = nodes[commonNodeIndex * 2];
                        commonNodey = nodes[commonNodeIndex * 2 + 1];
                    }
                    if (surSegmentCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += PENALITE_MAX_SELF; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateCrossingsReel(float* nodes, const int* edges, int* scores, float* newCoords, const int* nodeId, const int numThreads, const int numNodes, const int numEdges, const int* commonNodeEdges, const int gridWidth, const int gridHeight, const int PENALITE_MAX, const int PENALITE_MAX_SELF) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int id = nodeId[i];
    float newX, newY;
    hiprandState state;
    hiprand_init(clock(), i, 0, &state);
    if (i == 0) {
        newX = nodes[id *2];
        newY = nodes[id*2 + 1];
    }
    else {
        newX = hiprand_uniform(&state) * gridWidth;
        newY = hiprand_uniform(&state) * gridHeight;
        newCoords[i * 2] = newX;
        newCoords[i * 2 + 1] = newY;
    }
    int score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int areteIndex1, areteIndex2;
    float node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    float commonNodex, commonNodey;
    int limiteIndex1 = (numEdges * 2) - 2;
    int limiteIndex2 = (numEdges * 2);
    for (int i = 0; i < limiteIndex1; i += 2) {
        areteIndex1 = i / 2;
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1 * 2];
        node1y = nodes[nodeIndex1 * 2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2 * 2];
        node2y = nodes[nodeIndex2 * 2 + 1];
        if (nodeIndex1 == id) {
            node1x = newX;
            node1y = newY;
        }
        else if (nodeIndex2 == id) {
            node2x = newX;
            node2y = newY;
        }
        for (int j = i + 2; j < limiteIndex2; j += 2) {
            areteIndex2 = j / 2;
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3 * 2];
            node3y = nodes[nodeIndex3 * 2 + 1];
            nodeIndex4 = edges[j + 1];
            node4x = nodes[nodeIndex4 * 2];
            node4y = nodes[nodeIndex4 * 2 + 1];
            bool isIllegal = false;
            if (nodeIndex3 == id) {
                node3x = newX;
                node3y = newY;
            }
            else if (nodeIndex4 == id) {
                node4x = newX;
                node4y = newY;
            }
            if (commonNodeEdges[areteIndex1 * numEdges + areteIndex2] == -1) {
                if (seCroisentReelCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += PENALITE_MAX;  }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[areteIndex2 * numEdges + areteIndex1];
                if (commonNodeIndex == id) {
                    commonNodex = newX;
                    commonNodey = newY;
                }
                else {
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                }
                if (surSegmentReelCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += PENALITE_MAX_SELF; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[areteIndex1 * numEdges + areteIndex2];
                    if (commonNodeIndex == id) {
                        commonNodex = newX;
                        commonNodey = newY;
                    }
                    else {
                        commonNodex = nodes[commonNodeIndex * 2];
                        commonNodey = nodes[commonNodeIndex * 2 + 1];
                    }
                    if (surSegmentReelCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += PENALITE_MAX_SELF; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateArray(int* nodes, int* slots, int* newSlotPos, int* nodeId, int bestIndex) {
    int firstNodeId = nodeId[bestIndex];
    int newSlotId = newSlotPos[bestIndex];
    int oldSlotId = nodes[firstNodeId];
    if (slots[newSlotId] != -1) {
        int secondNodeId = slots[newSlotId];
        nodes[secondNodeId] = oldSlotId;
        nodes[firstNodeId] = newSlotId;
        slots[oldSlotId] = secondNodeId;
        slots[newSlotId] = firstNodeId;
    }
    else {
        nodes[firstNodeId] = newSlotId;
        slots[oldSlotId] = -1;
        slots[newSlotId] = firstNodeId;
    }
}

__global__ void kernelUpdateArrayReel(float* nodes, float* newCoords, int* nodeId, int bestIndex) {
    int id = nodeId[bestIndex] * 2;
    nodes[id] = newCoords[bestIndex * 2];
    nodes[id + 1] = newCoords[bestIndex * 2 + 1];
}

extern "C" void rechercheTabouGPU(int* nodes, const int* edges, const int* slots, const int* slotsCoord, int* scores, const int* newSlotPos, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int numSlots, const int blockSize, const int gridSize, const int placementScore, const int PENALITE_MAX, const int PENALITE_MAX_SELF) {
    int* devNodes;
    int* devEdges;
    int* devSlots;
    int* devSlotsCoord;
    int* devScores;
    int* devNewSlotPos;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(int) * numNodes);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devSlots, sizeof(int) * numSlots);
    hipMalloc((void**)&devSlotsCoord, sizeof(int) * numSlots * 2);
    hipMalloc((void**)&devScores, sizeof(int) * numThreads);
    hipMalloc((void**)&devNewSlotPos, sizeof(int) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);
    hipMalloc((void**)&devCommonNodeEdges, sizeof(int) * numEdges * numEdges);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(int) * numNodes, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devSlots, slots, sizeof(int) * numSlots, hipMemcpyHostToDevice);
    hipMemcpy(devSlotsCoord, slotsCoord, sizeof(int) * numSlots * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewSlotPos, newSlotPos, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devCommonNodeEdges, commonNodeEdges, sizeof(int) * numEdges * numEdges, hipMemcpyHostToDevice);

    int bestOverallScore = placementScore;
    for (int iter = 0; iter < 100; iter++) {
        // Update crossing score
        kernelUpdateCrossings<<<gridSize,blockSize>>>(devNodes, devEdges, devSlots, devSlotsCoord, devScores, devNewSlotPos, devNodeId, numThreads, numNodes, numEdges, numSlots, devCommonNodeEdges, PENALITE_MAX, PENALITE_MAX_SELF);
        // Update vector device to host
        hipMemcpy(scores, devScores, sizeof(int) * numThreads, hipMemcpyDeviceToHost);
        int bestValue = scores[0];
        int bestIndex = 0;
        for (int i = 1; i < numThreads; i++) {
            if (scores[i] < bestValue) {
                bestValue = scores[i];
                bestIndex = i;
            }
        }
        kernelUpdateArray<<<1,1>>>(devNodes, devSlots, devNewSlotPos, devNodeId, bestIndex);
        hipDeviceSynchronize();
        if (bestValue < bestOverallScore) {
            bestOverallScore = bestValue;
            hipMemcpy(nodes, devNodes, sizeof(int) * numNodes, hipMemcpyDeviceToHost);
        }
    }
    std::cout << "Best Score GPU: " << bestOverallScore << std::endl;
    //for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }
    //std::cout << "score: " << scores[0] << std::endl;
    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devSlots);
    hipFree(devSlotsCoord);
    hipFree(devScores);
    hipFree(devNewSlotPos);
    hipFree(devNodeId);
    hipFree(devCommonNodeEdges);
}

extern "C" void rechercheTabouGPUReel(float* nodes, const int* edges, int* scores,const float* newCoords, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int blockSize, const int gridSize, const int gridWidth, const int gridHeight, const int placementScore, const int PENALITE_MAX, const int PENALITE_MAX_SELF,const double timeLimit) {
    float* devNodes;
    int* devEdges;
    int* devScores;
    float* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(float) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(int) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(float) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);
    hipMalloc((void**)&devCommonNodeEdges, sizeof(int) * numEdges * numEdges);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(float) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(float) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devCommonNodeEdges, commonNodeEdges, sizeof(int) * numEdges * numEdges, hipMemcpyHostToDevice);
    auto start = std::chrono::system_clock::now();
    auto end = start;
    std::chrono::duration<double> secondsTotal = end - start;
    int bestOverallScore = placementScore;
    while(secondsTotal.count() < timeLimit) {
    //for (int iter = 0; iter < 10000; iter++) {
        // Update crossing score
        kernelUpdateCrossingsReel<<<gridSize, blockSize>>>(devNodes, devEdges, devScores, devNewCoords, devNodeId, numThreads, numNodes, numEdges, devCommonNodeEdges,gridWidth,gridHeight,PENALITE_MAX,PENALITE_MAX_SELF);
        // Update vector device to host
        hipMemcpy(scores, devScores, sizeof(int) * numThreads, hipMemcpyDeviceToHost);
        int bestValue = scores[0];
        int bestIndex = 0;
        for (int i = 1; i < numThreads; i++) {
            if (scores[i] < bestValue) {
                bestValue = scores[i];
                bestIndex = i;
            }
        }
        kernelUpdateArrayReel<<<1,1>>>(devNodes, devNewCoords, devNodeId, bestIndex);
        hipDeviceSynchronize();
        tcout() << "GPU score actuel: " << bestValue << std::endl;
        if (bestValue < bestOverallScore) {
            bestOverallScore = bestValue;
            hipMemcpy(nodes, devNodes, sizeof(float) * numNodes * 2, hipMemcpyDeviceToHost);
#if defined(DEBUG_GRAPHE_PROGRESS)
            tcout() << "GPU new best: " << bestOverallScore << std::endl;
#endif
        }
        end = std::chrono::system_clock::now();
        secondsTotal = end - start;
    }
    tcout() << "Best Score GPU: " << bestOverallScore << std::endl;
    //for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }
    //std::cout << "score: " << scores[0] << std::endl;
    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
    hipFree(devCommonNodeEdges);
}