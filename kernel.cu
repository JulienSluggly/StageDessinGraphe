#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include <ctime>
#include <chrono>
#include <iomanip>
#include <fstream>

#include "personnel.hpp"
#include "kernel.h"

__forceinline__ __device__ float diff_of_products(int a, int b, int c, int d) {
    int w = d * c;
    int e = fmaf(-d, c, w);
    int f = fmaf(a, b, -w);
    return f + e;
}

__device__ int area2CUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    return area2CUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool xorBoolCUDA(bool x, bool y) {
    return !x ^ !y;
}

__device__ bool intersectPropCUDA(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy) {
    if (collinearCUDA(ax, ay, bx, by, cx, cy) || collinearCUDA(ax, ay, bx, by, dx, dy) || collinearCUDA(cx, cy, dx, dy, ax, ay) || collinearCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftCUDA(ax, ay, bx, by, cx, cy), leftCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftCUDA(cx, cy, dx, dy, ax, ay), leftCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentCUDA(int ax, int ay, int bx, int by, int cx, int cy, int dx, int dy, bool& isIllegal) {
    if (intersectPropCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenCUDA(ax, ay, bx, by, cx, cy) || betweenCUDA(ax, ay, bx, by, dx, dy) || betweenCUDA(cx, cy, dx, dy, ax, ay) || betweenCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentCUDA(int ax, int ay, int bx, int by, int cx, int cy) {
    if (!collinearCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__forceinline__ __device__ float diff_of_productsReel(float a, float b, float c, float d) {
    float w = d * c;
    float e = fmaf(-d, c, w);
    float f = fmaf(a, b, -w);
    return f + e;
}

__device__ float area2ReelCUDA(float ax, float ay, float bx, float by, float cx, float cy) {
    //return diff_of_productsReel((bx - ax), (cy - ay), (cx - ax), (by - ay));
    return (bx - ax) * (cy - ay) - (cx - ax) * (by - ay);
}

__device__ bool leftReelCUDA(float ax, float ay, float bx, float by, float cx, float cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) > 0;
}

__device__ bool collinearReelCUDA(float ax, float ay, float bx, float by, float cx, float cy) {
    return area2ReelCUDA(ax, ay, bx, by, cx, cy) == 0;
}

__device__ bool intersectPropReelCUDA(float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy) {
    if (collinearReelCUDA(ax, ay, bx, by, cx, cy) || collinearReelCUDA(ax, ay, bx, by, dx, dy) || collinearReelCUDA(cx, cy, dx, dy, ax, ay) || collinearReelCUDA(cx, cy, dx, dy, bx, by))
        return false;
    return xorBoolCUDA(leftReelCUDA(ax, ay, bx, by, cx, cy), leftReelCUDA(ax, ay, bx, by, dx, dy)) && xorBoolCUDA(leftReelCUDA(cx, cy, dx, dy, ax, ay), leftReelCUDA(cx, cy, dx, dy, bx, by));
}

__device__ bool betweenReelCUDA(float ax, float ay, float bx, float by, float cx, float cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__device__ bool seCroisentReelCUDA(float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy, bool& isIllegal) {
    if (intersectPropReelCUDA(ax, ay, bx, by, cx, cy, dx, dy))
        return true;
    else if (betweenReelCUDA(ax, ay, bx, by, cx, cy) || betweenReelCUDA(ax, ay, bx, by, dx, dy) || betweenReelCUDA(cx, cy, dx, dy, ax, ay) || betweenReelCUDA(cx, cy, dx, dy, bx, by)) {
        isIllegal = true;
        return true;
    }
    return false;
}

__device__ bool surSegmentReelCUDA(float ax, float ay, float bx, float by, float cx, float cy) {
    if (!collinearReelCUDA(ax, ay, bx, by, cx, cy))
        return false;
    if (ax != bx)
        return ((ax <= cx) && (cx <= bx)) || ((ax >= cx) && (cx >= bx));
    else
        return ((ay <= cy) && (cy <= by) || ((ay >= cy) && (cy >= by)));
}

__global__ void kernelUpdateCrossings(int* nodes, int* edges, int* scores, int* newCoords, int* nodeId, int numThreads,int numNodes, int numEdges, int* commonNodeEdges, int gridWidth, int gridHeight) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > numThreads) { return; }
    int id = nodeId[i]*2;
    int newX = newCoords[i*2];
    int newY = newCoords[i*2+1];
    int score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    int commonNodex, commonNodey;
    for (int i = 0; i < numEdges-1; i+=2) {
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1*2];
        node1y = nodes[nodeIndex1*2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2*2];
        node2y = nodes[nodeIndex2*2 + 1];
        for (int j = i + 1; j < numEdges; j+=2) {
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3*2];
            node3y = nodes[nodeIndex3*2 + 1];
            nodeIndex4 = edges[j+1];
            node4x = nodes[nodeIndex4*2];
            node4y = nodes[nodeIndex4*2 + 1];
            bool isIllegal = false;
            if (commonNodeEdges[i*numNodes+j] == -1) {
                if (seCroisentCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000; }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[j*numNodes+i];
                commonNodex = nodes[commonNodeIndex * 2];
                commonNodey = nodes[commonNodeIndex * 2 + 1];
                if (surSegmentCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[i*numNodes+j];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegmentCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateCrossingsReel(float* nodes, int* edges, int* scores, float* newCoords, int* nodeId, int numThreads, int numNodes, int numEdges, int* commonNodeEdges, int gridWidth, int gridHeight) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int id = nodeId[i];
    float newX, newY;
    hiprandState state;
    hiprand_init(clock(), i, 0, &state);
    if (i == 0) {
        newX = nodes[id *2];
        newY = nodes[id*2 + 1];
    }
    else {
        newX = hiprand_uniform(&state) * gridWidth;
        newY = hiprand_uniform(&state) * gridHeight;
        newCoords[i * 2] = newX;
        newCoords[i * 2 + 1] = newY;
    }
    int score = 0;
    int nodeIndex1, nodeIndex2, nodeIndex3, nodeIndex4;
    int areteIndex1, areteIndex2;
    float node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y;
    int commonNodeIndex;
    float commonNodex, commonNodey;
    int limiteIndex1 = (numEdges * 2) - 2;
    int limiteIndex2 = (numEdges * 2);
    for (int i = 0; i < limiteIndex1; i += 2) {
        areteIndex1 = i / 2;
        nodeIndex1 = edges[i];
        node1x = nodes[nodeIndex1 * 2];
        node1y = nodes[nodeIndex1 * 2 + 1];
        nodeIndex2 = edges[i + 1];
        node2x = nodes[nodeIndex2 * 2];
        node2y = nodes[nodeIndex2 * 2 + 1];
        if (nodeIndex1 == id) {
            node1x = newX;
            node1y = newY;
        }
        else if (nodeIndex2 == id) {
            node2x = newX;
            node2y = newY;
        }
        for (int j = i + 2; j < limiteIndex2; j += 2) {
            areteIndex2 = j / 2;
            nodeIndex3 = edges[j];
            node3x = nodes[nodeIndex3 * 2];
            node3y = nodes[nodeIndex3 * 2 + 1];
            nodeIndex4 = edges[j + 1];
            node4x = nodes[nodeIndex4 * 2];
            node4y = nodes[nodeIndex4 * 2 + 1];
            bool isIllegal = false;
            if (nodeIndex3 == id) {
                node3x = newX;
                node3y = newY;
            }
            else if (nodeIndex4 == id) {
                node4x = newX;
                node4y = newY;
            }
            if (commonNodeEdges[areteIndex1 * numEdges + areteIndex2] == -1) {
                if (seCroisentReelCUDA(node1x, node1y, node2x, node2y, node3x, node3y, node4x, node4y, isIllegal)) {
                    if (isIllegal) { score += 1000;  }
                    else { score++; }
                }
            }
            else {
                commonNodeIndex = commonNodeEdges[areteIndex2 * numEdges + areteIndex1];
                if (commonNodeIndex == id) {
                    commonNodex = newX;
                    commonNodey = newY;
                }
                else {
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                }
                if (surSegmentReelCUDA(node1x, node1y, node2x, node2y, commonNodex, commonNodey)) { score += 1001; isIllegal = true; }
                if (!isIllegal) {
                    commonNodeIndex = commonNodeEdges[areteIndex1 * numEdges + areteIndex2];
                    commonNodex = nodes[commonNodeIndex * 2];
                    commonNodey = nodes[commonNodeIndex * 2 + 1];
                    if (surSegmentReelCUDA(node3x, node3y, node4x, node4y, commonNodex, commonNodey)) { score += 1001; }
                }
            }
        }
    }
    scores[i] = score;
}

__global__ void kernelUpdateArray(float* nodes, float* newCoords, int* nodeId, int bestIndex) {
    int id = nodeId[bestIndex] * 2;
    nodes[id] = newCoords[bestIndex * 2];
    nodes[id + 1] = newCoords[bestIndex * 2 + 1];
}

extern "C" void rechercheTabouGPU(int* nodes, const int* edges, int* scores,const int* newCoords, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int blockSize, const int gridSize, const int gridWidth, const int gridHeight) {
    int* devNodes;
    int* devEdges;
    int* devScores;
    int* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(int) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(int) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(int) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(int) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(int) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);

    // Update crossing score
    kernelUpdateCrossings<<<gridSize, blockSize>>>(devNodes,devEdges,devScores,devNewCoords,devNodeId,numThreads,numNodes,numEdges,devCommonNodeEdges,gridWidth,gridHeight);
    // Update vector device to host
    hipMemcpy(scores, devScores, sizeof(int) * numThreads, hipMemcpyDeviceToHost);

    for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }

    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
}

extern "C" void rechercheTabouGPUReel(float* nodes, const int* edges, int* scores,const float* newCoords, const int* nodeId, const int* commonNodeEdges, const int numNodes, const int numEdges, const int blockSize, const int gridSize, const int gridWidth, const int gridHeight, const int placementScore) {
    float* devNodes;
    int* devEdges;
    int* devScores;
    float* devNewCoords;
    int* devNodeId;
    int* devCommonNodeEdges;

    int numThreads = blockSize * gridSize;

    // Memory allocation
    hipMalloc((void**)&devNodes, sizeof(float) * numNodes * 2);
    hipMalloc((void**)&devEdges, sizeof(int) * numEdges * 2);
    hipMalloc((void**)&devScores, sizeof(int) * numThreads);
    hipMalloc((void**)&devNewCoords, sizeof(float) * numThreads * 2);
    hipMalloc((void**)&devNodeId, sizeof(int) * numThreads);
    hipMalloc((void**)&devCommonNodeEdges, sizeof(int) * numEdges * numEdges);

    // Copy vectors datas from host to device
    hipMemcpy(devNodes, nodes, sizeof(float) * numNodes * 2, hipMemcpyHostToDevice);
    hipMemcpy(devEdges, edges, sizeof(int) * numEdges * 2, hipMemcpyHostToDevice);
    hipMemcpy(devScores, scores, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devNewCoords, newCoords, sizeof(float) * numThreads * 2, hipMemcpyHostToDevice);
    hipMemcpy(devNodeId, nodeId, sizeof(int) * numThreads, hipMemcpyHostToDevice);
    hipMemcpy(devCommonNodeEdges, commonNodeEdges, sizeof(int) * numEdges * numEdges, hipMemcpyHostToDevice);

    int bestOverallScore = placementScore;
    for (int iter = 0; iter < 10000; iter++) {
        // Update crossing score
        kernelUpdateCrossingsReel<<<gridSize, blockSize>>>(devNodes, devEdges, devScores, devNewCoords, devNodeId, numThreads, numNodes, numEdges, devCommonNodeEdges,gridWidth,gridHeight);
        // Update vector device to host
        hipMemcpy(scores, devScores, sizeof(int) * numThreads, hipMemcpyDeviceToHost);
        int bestValue = scores[0];
        int bestIndex = 0;
        for (int i = 1; i < numThreads; i++) {
            if (scores[i] < bestValue) {
                bestValue = scores[i];
                bestIndex = i;
            }
        }
        kernelUpdateArray<<<1,1>>>(devNodes, devNewCoords, devNodeId, bestIndex);
        hipDeviceSynchronize();
        if (bestValue < bestOverallScore) {
            bestOverallScore = bestValue;
            hipMemcpy(nodes, devNodes, sizeof(float) * numNodes * 2, hipMemcpyDeviceToHost);
        }
    }
    std::cout << "Best Score GPU: " << bestOverallScore << std::endl;
    //for (int i = 0; i < numThreads; i++) { std::cout << "i: " << i << " " << scores[i] << std::endl; }
    //std::cout << "score: " << scores[0] << std::endl;
    // cleanup
    hipFree(devNodes);
    hipFree(devEdges);
    hipFree(devScores);
    hipFree(devNewCoords);
    hipFree(devNodeId);
    hipFree(devCommonNodeEdges);
}